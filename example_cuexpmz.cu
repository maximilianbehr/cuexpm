/* MIT License
 *
 * Copyright (c) 2024 Maximilian Behr
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <chrono>
#include <cstdio>
#include <cstdlib>

#include "cuexpm.h"

int main(void) {
    /*-----------------------------------------------------------------------------
     * variables
     *-----------------------------------------------------------------------------*/
    int ret = 0;
    int n = 1 << 10;  // size of the matrix

    hipDoubleComplex *A, *expmA;      // A and expmA on the host
    hipDoubleComplex *d_A, *d_expmA;  // A and expmA on the device
    void *d_buffer = NULL;           // memory buffer on the device
    void *h_buffer = NULL;           // memory buffer on the host

    /*------------------------------------------------------------------------------
     * allocate A and expmA on the host
     *-----------------------------------------------------------------------------*/
    hipHostMalloc((void **)&A, sizeof(*A) * n * n);
    hipHostMalloc((void **)&expmA, sizeof(*expmA) * n * n);

    /*-----------------------------------------------------------------------------
     * fill matrix A
     *-----------------------------------------------------------------------------*/
    for (int j = 0; j < n; ++j) {
        for (int i = 0; i < n; ++i) {
            if (i >= j) {
                A[i + j * n] = {-.1, -.1};
            } else {
                A[i + j * n] = {-.3, -.1};
            }
        }
    }

    /*-----------------------------------------------------------------------------
     * copy A to the decive
     *-----------------------------------------------------------------------------*/
    hipMalloc((void **)&d_A, sizeof(*d_A) * n * n);
    hipMemcpy(d_A, A, sizeof(*A) * n * n, hipMemcpyHostToDevice);

    /*-----------------------------------------------------------------------------
     * allocate expmA on the device
     *-----------------------------------------------------------------------------*/
    hipMalloc((void **)&d_expmA, sizeof(*d_expmA) * n * n);

    /*-----------------------------------------------------------------------------
     * workspace query and allocate memory buffer on the device and the host
     *-----------------------------------------------------------------------------*/
    size_t d_bufferSize = 0, h_bufferSize = 0;
    ret = cuexpmz_bufferSize(n, &d_bufferSize, &h_bufferSize);
    if (ret) {
        fprintf(stderr, "cuexpmz_bufferSize failed with error %d\n", ret);
        fflush(stderr);
        return ret;
    }
    if (d_bufferSize > 0) {
        hipMalloc((void **)&d_buffer, d_bufferSize);
    }
    if (h_bufferSize > 0) {
        hipHostMalloc((void **)&h_buffer, h_bufferSize);
    }

    /*-----------------------------------------------------------------------------
     * compute the approximation of the matrix exponential of A and measure the time
     *-----------------------------------------------------------------------------*/
    auto t0 = std::chrono::high_resolution_clock::now();
    ret = cuexpmz(d_A, n, d_buffer, h_buffer, d_expmA);
    if (ret) {
        fprintf(stderr, "cuexpmz failed with error %d\n", ret);
        fflush(stderr);
        return ret;
    }
    auto t1 = std::chrono::high_resolution_clock::now();
    double wtime = std::chrono::duration_cast<std::chrono::nanoseconds>(t1 - t0).count();

    /*-----------------------------------------------------------------------------
     * copy result to host and print the first 5x5 block
     *-----------------------------------------------------------------------------*/
    hipMemcpy(expmA, d_expmA, sizeof(*d_expmA) * n * n, hipMemcpyDeviceToHost);

    printf("expmA(1:5, 1:5) =\n");
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%+e%+e*i ", expmA[i + j * n].x, expmA[i + j * n].y);
        }
        printf("\n");
    }
    printf("WallClockTime = %fs\n", wtime * 1e-9);

    /*-----------------------------------------------------------------------------
     * clear matrices A, expmA, d_A, and d_expmA and the device and host buffer
     *-----------------------------------------------------------------------------*/
    hipHostFree(A);
    hipHostFree(expmA);
    hipFree(d_A);
    hipFree(d_expmA);
    hipFree(d_buffer);
    hipHostFree(h_buffer);
    return 0;
}
